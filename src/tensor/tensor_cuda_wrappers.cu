#include "hip/hip_runtime.h"
#include "../../inc/tensor/tensor_cuda_wrappers.hpp"
#include "tensor_cuda_kernels.cu"

namespace cuda_ops {

// Memory operations
template<typename T>
T* cuda_malloc(size_t size) {
    T* ptr;
    CUDA_CHECK(hipMalloc(&ptr, size * sizeof(T)));
    return ptr;
}

template<typename T>
void cuda_free(T* ptr) {
    CUDA_CHECK(hipFree(ptr));
}

template<typename T>
void cuda_memcpy_host_to_device(T* dst, const T* src, size_t size) {
    CUDA_CHECK(hipMemcpy(dst, src, size * sizeof(T), hipMemcpyHostToDevice));
}

template<typename T>
void cuda_memcpy_device_to_host(T* dst, const T* src, size_t size) {
    CUDA_CHECK(hipMemcpy(dst, src, size * sizeof(T), hipMemcpyDeviceToHost));
}

template<typename T>
void cuda_memcpy_device_to_device(T* dst, const T* src, size_t size) {
    CUDA_CHECK(hipMemcpy(dst, src, size * sizeof(T), hipMemcpyDeviceToDevice));
}

// Element-wise operations
template<typename T, typename U, typename R>
void launch_tensor_add(const T* a, const U* b, R* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);

    auto op = [] __device__ (const T& x, const U& y) -> R { return static_cast<R>(x) + static_cast<R>(y); };
    tensor_elementwise_kernel<<<grid, block>>>(a, b, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U, typename R>
void launch_tensor_subtract(const T* a, const U* b, R* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);

    auto op = [] __device__ (const T& x, const U& y) -> R { return static_cast<R>(x) - static_cast<R>(y); };
    tensor_elementwise_kernel<<<grid, block>>>(a, b, result, size, op);
    
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U, typename R>
void launch_tensor_multiply(const T* a, const U* b, R* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);

    auto op = [] __device__ (const T& x, const U& y) -> R { return static_cast<R>(x) * static_cast<R>(y); };
    tensor_elementwise_kernel<<<grid, block>>>(a, b, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U, typename R>
void launch_tensor_divide(const T* a, const U* b, R* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> R { return static_cast<R>(x) / static_cast<R>(y); };
    tensor_elementwise_kernel<<<grid, block>>>(a, b, result, size, op);
    
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// Matrix multiplication
template<typename T, typename U, typename R>
void launch_tensor_matmul(const T* a, const U* b, R* result, int M, int N, int K) {
    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);
    
    tensor_matmul_kernel<<<grid, block>>>(a, b, result, M, N, K);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// Copy
template<typename T>
void launch_tensor_copy(const T* src, T* dst, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    tensor_copy_kernel<<<grid, block>>>(src, dst, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// Fill
template<typename T>
void launch_tensor_fill(T* data, T value, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    tensor_fill_kernel<<<grid, block>>>(data, value, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// Scalar operations
template<typename T, typename U, typename R>
void launch_tensor_scalar_add(const T* a, U scalar, R* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> R { return static_cast<R>(x) + static_cast<R>(y); };
    tensor_scalar_kernel<<<grid, block>>>(a, scalar, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U, typename R>
void launch_tensor_scalar_subtract(const T* a, U scalar, R* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> R { return static_cast<R>(x) - static_cast<R>(y); };
    tensor_scalar_kernel<<<grid, block>>>(a, scalar, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U, typename R>
void launch_tensor_scalar_multiply(const T* a, U scalar, R* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> R { return static_cast<R>(x) * static_cast<R>(y); };
    tensor_scalar_kernel<<<grid, block>>>(a, scalar, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U, typename R>
void launch_tensor_scalar_divide(const T* a, U scalar, R* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> R { return static_cast<R>(x) / static_cast<R>(y); };
    tensor_scalar_kernel<<<grid, block>>>(a, scalar, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// In-place operations
template<typename T, typename U>
void launch_tensor_inplace_add(T* a, const U* b, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (T& x, const U& y) { x += static_cast<T>(y); };
    tensor_inplace_elementwise_kernel<<<grid, block>>>(a, b, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_inplace_subtract(T* a, const U* b, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);

    auto op = [] __device__ (T& x, const U& y) { x -= static_cast<T>(y); };
    tensor_inplace_elementwise_kernel<<<grid, block>>>(a, b, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_inplace_multiply(T* a, const U* b, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (T& x, const U& y) { x *= static_cast<T>(y); };
    tensor_inplace_elementwise_kernel<<<grid, block>>>(a, b, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_inplace_divide(T* a, const U* b, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (T& x, const U& y) { x /= static_cast<T>(y); };
    tensor_inplace_elementwise_kernel<<<grid, block>>>(a, b, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// In-place scalar operations
template<typename T, typename U>
void launch_tensor_inplace_scalar_add(T* a, U scalar, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (T& x, const U& y) { x += static_cast<T>(y); };
    tensor_inplace_scalar_kernel<<<grid, block>>>(a, scalar, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_inplace_scalar_subtract(T* a, U scalar, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (T& x, const U& y) { x -= static_cast<T>(y); };
    tensor_inplace_scalar_kernel<<<grid, block>>>(a, scalar, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_inplace_scalar_multiply(T* a, U scalar, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (T& x, const U& y) { x *= static_cast<T>(y); };
    tensor_inplace_scalar_kernel<<<grid, block>>>(a, scalar, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_inplace_scalar_divide(T* a, U scalar, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (T& x, const U& y) { x /= static_cast<T>(y); };
    tensor_inplace_scalar_kernel<<<grid, block>>>(a, scalar, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// Element-wise operations
template<typename T>
void launch_tensor_abs(const T* a, T* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    tensor_abs_kernel<<<grid, block>>>(a, result, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T>
void launch_tensor_pow(const T* a, T* result, double exponent, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    tensor_pow_kernel<<<grid, block>>>(a, result, exponent, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T>
void launch_tensor_normalize(const T* a, T* result, T min_val, T max_val, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    tensor_normalize_kernel<<<grid, block>>>(a, result, min_val, max_val, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// Matrix operations
template<typename T>
void launch_tensor_transpose(const T* a, T* result, int rows, int cols) {
    dim3 block(16, 16);
    dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
    
    tensor_transpose_kernel<<<grid, block>>>(a, result, rows, cols);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T>
void launch_tensor_flip(const T* a, T* result, const int* shape, const int* axes,
                       const int* strides, const int* result_strides,
                       int ndim, int num_axes, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    tensor_flip_kernel<<<grid, block>>>(a, result, shape, axes, strides, result_strides, ndim, num_axes, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// Reduction operations
template<typename T>
T launch_tensor_min(const T* data, size_t size) {
    const int block_size = 256;
    const int grid_size = (size + block_size - 1) / block_size;
    
    T* d_result;
    CUDA_CHECK(hipMalloc(&d_result, grid_size * sizeof(T)));
    
    tensor_min_kernel<<<grid_size, block_size, block_size * sizeof(T)>>>(data, d_result, size);
    CUDA_CHECK(hipGetLastError());
    
    if(grid_size > 1) {
        T result = launch_tensor_min(d_result, grid_size);
        CUDA_CHECK(hipFree(d_result));
        return result;
    } else {
        T result;
        CUDA_CHECK(hipMemcpy(&result, d_result, sizeof(T), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(d_result));
        return result;
    }
}

template<typename T>
T launch_tensor_max(const T* data, size_t size) {
    const int block_size = 256;
    const int grid_size = (size + block_size - 1) / block_size;
    
    T* d_result;
    CUDA_CHECK(hipMalloc(&d_result, grid_size * sizeof(T)));
    
    tensor_max_kernel<<<grid_size, block_size, block_size * sizeof(T)>>>(data, d_result, size);
    CUDA_CHECK(hipGetLastError());
    
    if(grid_size > 1) {
        T result = launch_tensor_max(d_result, grid_size);
        CUDA_CHECK(hipFree(d_result));
        return result;
    } else {
        T result;
        CUDA_CHECK(hipMemcpy(&result, d_result, sizeof(T), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(d_result));
        return result;
    }
}

template<typename T>
T launch_tensor_sum(const T* data, size_t size) {
    const int block_size = 256;
    const int grid_size = (size + block_size - 1) / block_size;
    
    T* d_result;
    CUDA_CHECK(hipMalloc(&d_result, grid_size * sizeof(T)));
    
    tensor_sum_kernel<<<grid_size, block_size, block_size * sizeof(T)>>>(data, d_result, size);
    CUDA_CHECK(hipGetLastError());
    
    if(grid_size > 1) {
        T result = launch_tensor_sum(d_result, grid_size);
        CUDA_CHECK(hipFree(d_result));
        return result;
    } else {
        T result;
        CUDA_CHECK(hipMemcpy(&result, d_result, sizeof(T), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(d_result));
        return result;
    }
}

template<typename T>
T launch_tensor_norm(const T* data, size_t size) {
    const int block_size = 256;
    const int grid_size = (size + block_size - 1) / block_size;
    
    T* d_result;
    CUDA_CHECK(hipMalloc(&d_result, grid_size * sizeof(T)));
    
    tensor_norm_squared_kernel<<<grid_size, block_size, block_size * sizeof(T)>>>(data, d_result, size);
    CUDA_CHECK(hipGetLastError());
    
    T sum_of_squares;
    if(grid_size > 1) {
        sum_of_squares = launch_tensor_sum(d_result, grid_size);
    } else {
        CUDA_CHECK(hipMemcpy(&sum_of_squares, d_result, sizeof(T), hipMemcpyDeviceToHost));
    }
    
    CUDA_CHECK(hipFree(d_result));
    return std::sqrt(sum_of_squares);
}

// New reduction operations
template<typename T>
T launch_tensor_variance(const T* data, T mean_val, size_t size) {
    const int block_size = 256;
    const int grid_size = (size + block_size - 1) / block_size;
    
    T* d_result;
    CUDA_CHECK(hipMalloc(&d_result, grid_size * sizeof(T)));
    
    tensor_variance_kernel<<<grid_size, block_size, block_size * sizeof(T)>>>(data, mean_val, d_result, size);
    CUDA_CHECK(hipGetLastError());
    
    T* h_result = new T[grid_size];
    CUDA_CHECK(hipMemcpy(h_result, d_result, grid_size * sizeof(T), hipMemcpyDeviceToHost));
    
    T final_var = T(0);
    for(int i = 0; i < grid_size; i++) {
        final_var += h_result[i];
    }
    
    delete[] h_result;
    CUDA_CHECK(hipFree(d_result));
    
    return final_var / T(size - 1);  // Population variance to match CPU
}

// Slice operation
template<typename T>
void launch_tensor_slice(const T* data, T* result, int start, int step, size_t new_size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(new_size, block_size);
    dim3 block(block_size);
    
    tensor_slice_kernel<<<grid, block>>>(data, result, start, step, new_size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// Square root operation
template<typename T>
void launch_tensor_sqrt(const T* data, T* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    tensor_sqrt_kernel<<<grid, block>>>(data, result, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// Boolean operations
template<typename T, typename U>
void launch_tensor_equal(const T* a, const U* b, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x == y; };
    tensor_elementwise_kernel<<<grid, block>>>(a, b, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_not_equal(const T* a, const U* b, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x != y; };
    tensor_elementwise_kernel<<<grid, block>>>(a, b, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_less_than(const T* a, const U* b, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x < y; };
    tensor_elementwise_kernel<<<grid, block>>>(a, b, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_less_equal(const T* a, const U* b, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x <= y; };
    tensor_elementwise_kernel<<<grid, block>>>(a, b, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_greater_than(const T* a, const U* b, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x > y; };
    tensor_elementwise_kernel<<<grid, block>>>(a, b, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_greater_equal(const T* a, const U* b, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x >= y; };
    tensor_elementwise_kernel<<<grid, block>>>(a, b, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// Scalar boolean operations
template<typename T, typename U>
void launch_tensor_scalar_equal(const T* a, U scalar, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x == y; };
    tensor_scalar_kernel<<<grid, block>>>(a, scalar, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_scalar_not_equal(const T* a, U scalar, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x != y; };
    tensor_scalar_kernel<<<grid, block>>>(a, scalar, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_scalar_less_than(const T* a, U scalar, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x < y; };
    tensor_scalar_kernel<<<grid, block>>>(a, scalar, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_scalar_less_equal(const T* a, U scalar, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x <= y; };
    tensor_scalar_kernel<<<grid, block>>>(a, scalar, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_scalar_greater_than(const T* a, U scalar, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x > y; };
    tensor_scalar_kernel<<<grid, block>>>(a, scalar, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_scalar_greater_equal(const T* a, U scalar, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x >= y; };
    tensor_scalar_kernel<<<grid, block>>>(a, scalar, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// Logical operations
template<typename T, typename U>
void launch_tensor_logical_and(const T* a, const U* b, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x && y; };
    tensor_elementwise_kernel<<<grid, block>>>(a, b, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_logical_or(const T* a, const U* b, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x || y; };
    tensor_elementwise_kernel<<<grid, block>>>(a, b, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_scalar_logical_and(const T* a, U scalar, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x && y; };
    tensor_scalar_kernel<<<grid, block>>>(a, scalar, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T, typename U>
void launch_tensor_scalar_logical_or(const T* a, U scalar, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    auto op = [] __device__ (const T& x, const U& y) -> bool { return x || y; };
    tensor_scalar_kernel<<<grid, block>>>(a, scalar, result, size, op);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T>
void launch_tensor_logical_not(const T* a, bool* result, size_t size) {
    const int block_size = 256;
    dim3 grid = calculate_grid_block(size, block_size);
    dim3 block(block_size);
    
    tensor_logical_not_kernel<<<grid, block>>>(a, result, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

template<typename T>
bool launch_tensor_any(const T* data, size_t size) {
    const int block_size = 256;
    const int grid_size = (size + block_size - 1) / block_size;
    
    bool* d_result;
    CUDA_CHECK(hipMalloc(&d_result, grid_size * sizeof(bool)));
    
    tensor_any_kernel<<<grid_size, block_size, block_size * sizeof(bool)>>>(data, d_result, size);
    CUDA_CHECK(hipGetLastError());
    
    bool* h_result = new bool[grid_size];
    CUDA_CHECK(hipMemcpy(h_result, d_result, grid_size * sizeof(bool), hipMemcpyDeviceToHost));
    
    bool final_result = false;
    for(int i = 0; i < grid_size; i++) {
        if(h_result[i]) {
            final_result = true;
            break;
        }
    }
    
    delete[] h_result;
    CUDA_CHECK(hipFree(d_result));
    
    return final_result;
}

template<typename T>
bool launch_tensor_all(const T* data, size_t size) {
    const int block_size = 256;
    const int grid_size = (size + block_size - 1) / block_size;
    
    bool* d_result;
    CUDA_CHECK(hipMalloc(&d_result, grid_size * sizeof(bool)));
    
    tensor_all_kernel<<<grid_size, block_size, block_size * sizeof(bool)>>>(data, d_result, size);
    CUDA_CHECK(hipGetLastError());
    
    bool* h_result = new bool[grid_size];
    CUDA_CHECK(hipMemcpy(h_result, d_result, grid_size * sizeof(bool), hipMemcpyDeviceToHost));
    
    bool final_result = true;
    for(int i = 0; i < grid_size; i++) {
        if(!h_result[i]) {
            final_result = false;
            break;
        }
    }
    
    delete[] h_result;
    CUDA_CHECK(hipFree(d_result));
    
    return final_result;
}

// Explicit instantiations
template float* cuda_malloc<float>(size_t);
template void cuda_free<float>(float*);
template void cuda_memcpy_host_to_device<float>(float*, const float*, size_t);
template void cuda_memcpy_device_to_host<float>(float*, const float*, size_t);
template void cuda_memcpy_device_to_device<float>(float*, const float*, size_t);

template int* cuda_malloc<int>(size_t);
template void cuda_free<int>(int*);
template void cuda_memcpy_host_to_device<int>(int*, const int*, size_t);
template void cuda_memcpy_device_to_host<int>(int*, const int*, size_t);
template void cuda_memcpy_device_to_device<int>(int*, const int*, size_t);

template double* cuda_malloc<double>(size_t);
template void cuda_free<double>(double*);
template void cuda_memcpy_host_to_device<double>(double*, const double*, size_t);
template void cuda_memcpy_device_to_host<double>(double*, const double*, size_t);
template void cuda_memcpy_device_to_device<double>(double*, const double*, size_t);

template bool* cuda_malloc<bool>(size_t);
template void cuda_free<bool>(bool*);
template void cuda_memcpy_host_to_device<bool>(bool*, const bool*, size_t);
template void cuda_memcpy_device_to_host<bool>(bool*, const bool*, size_t);
template void cuda_memcpy_device_to_device<bool>(bool*, const bool*, size_t);

// Arithmetic operations
template void launch_tensor_add<float, float, float>(const float*, const float*, float*, size_t);
template void launch_tensor_subtract<float, float, float>(const float*, const float*, float*, size_t);
template void launch_tensor_multiply<float, float, float>(const float*, const float*, float*, size_t);
template void launch_tensor_divide<float, float, float>(const float*, const float*, float*, size_t);
template void launch_tensor_matmul<float, float, float>(const float*, const float*, float*, int, int, int);
template void launch_tensor_copy<float>(const float*, float*, size_t);
template void launch_tensor_fill<float>(float*, float, size_t);

// Cross-type operations
template void launch_tensor_add<int, float, float>(const int*, const float*, float*, size_t);

// Operations for int
template void launch_tensor_copy<int>(const int*, int*, size_t);
template void launch_tensor_fill<int>(int*, int, size_t);

// Operations for double  
template void launch_tensor_copy<double>(const double*, double*, size_t);
template void launch_tensor_fill<double>(double*, double, size_t);

// Scalar operations
template void launch_tensor_scalar_add<float, float, float>(const float*, float, float*, size_t);
template void launch_tensor_scalar_subtract<float, float, float>(const float*, float, float*, size_t);
template void launch_tensor_scalar_multiply<float, float, float>(const float*, float, float*, size_t);
template void launch_tensor_scalar_divide<float, float, float>(const float*, float, float*, size_t);

// In-place operations
template void launch_tensor_inplace_add<float, float>(float*, const float*, size_t);
template void launch_tensor_inplace_subtract<float, float>(float*, const float*, size_t);
template void launch_tensor_inplace_multiply<float, float>(float*, const float*, size_t);
template void launch_tensor_inplace_divide<float, float>(float*, const float*, size_t);

template void launch_tensor_inplace_scalar_add<float, float>(float*, float, size_t);
template void launch_tensor_inplace_scalar_subtract<float, float>(float*, float, size_t);
template void launch_tensor_inplace_scalar_multiply<float, float>(float*, float, size_t);
template void launch_tensor_inplace_scalar_divide<float, float>(float*, float, size_t);

// Reduction operations
template float launch_tensor_min<float>(const float*, size_t);
template float launch_tensor_max<float>(const float*, size_t);
template float launch_tensor_sum<float>(const float*, size_t);
template float launch_tensor_norm<float>(const float*, size_t);

// Boolean operations
template void launch_tensor_equal<float, float>(const float*, const float*, bool*, size_t);
template void launch_tensor_not_equal<float, float>(const float*, const float*, bool*, size_t);
template void launch_tensor_less_than<float, float>(const float*, const float*, bool*, size_t);
template void launch_tensor_less_equal<float, float>(const float*, const float*, bool*, size_t);
template void launch_tensor_greater_than<float, float>(const float*, const float*, bool*, size_t);
template void launch_tensor_greater_equal<float, float>(const float*, const float*, bool*, size_t);

template void launch_tensor_scalar_equal<float, float>(const float*, float, bool*, size_t);
template void launch_tensor_scalar_not_equal<float, float>(const float*, float, bool*, size_t);
template void launch_tensor_scalar_less_than<float, float>(const float*, float, bool*, size_t);
template void launch_tensor_scalar_less_equal<float, float>(const float*, float, bool*, size_t);
template void launch_tensor_scalar_greater_than<float, float>(const float*, float, bool*, size_t);
template void launch_tensor_scalar_greater_equal<float, float>(const float*, float, bool*, size_t);

template void launch_tensor_logical_and<float, float>(const float*, const float*, bool*, size_t);
template void launch_tensor_logical_or<float, float>(const float*, const float*, bool*, size_t);
template void launch_tensor_scalar_logical_and<float, float>(const float*, float, bool*, size_t);
template void launch_tensor_scalar_logical_or<float, float>(const float*, float, bool*, size_t);
template void launch_tensor_logical_not<float>(const float*, bool*, size_t);
template bool launch_tensor_any<float>(const float*, size_t);
template bool launch_tensor_all<float>(const float*, size_t);

template void launch_tensor_logical_and<bool, bool>(const bool*, const bool*, bool*, size_t);
template void launch_tensor_logical_or<bool, bool>(const bool*, const bool*, bool*, size_t);
template void launch_tensor_scalar_logical_and<bool, bool>(const bool*, bool, bool*, size_t);
template void launch_tensor_scalar_logical_or<bool, bool>(const bool*, bool, bool*, size_t);
template void launch_tensor_logical_not<bool>(const bool*, bool*, size_t);
template bool launch_tensor_any<bool>(const bool*, size_t);
template bool launch_tensor_all<bool>(const bool*, size_t);

// New operations
template float launch_tensor_variance<float>(const float*, float, size_t);
template void launch_tensor_slice<float>(const float*, float*, int, int, size_t);
template void launch_tensor_sqrt<float>(const float*, float*, size_t);

// Element-wise operations
template void launch_tensor_abs<float>(const float*, float*, size_t);
template void launch_tensor_abs<int>(const int*, int*, size_t);
template void launch_tensor_abs<double>(const double*, double*, size_t);

template void launch_tensor_pow<float>(const float*, float*, double, size_t);
template void launch_tensor_pow<int>(const int*, int*, double, size_t);
template void launch_tensor_pow<double>(const double*, double*, double, size_t);

template void launch_tensor_normalize<float>(const float*, float*, float, float, size_t);
template void launch_tensor_normalize<int>(const int*, int*, int, int, size_t);
template void launch_tensor_normalize<double>(const double*, double*, double, double, size_t);

// Matrix operations
template void launch_tensor_transpose<float>(const float*, float*, int, int);
template void launch_tensor_transpose<int>(const int*, int*, int, int);
template void launch_tensor_transpose<double>(const double*, double*, int, int);

template void launch_tensor_flip<float>(const float*, float*, const int*, const int*, const int*, const int*, int, int, size_t);
template void launch_tensor_flip<int>(const int*, int*, const int*, const int*, const int*, const int*, int, int, size_t);
template void launch_tensor_flip<double>(const double*, double*, const int*, const int*, const int*, const int*, int, int, size_t);

}
